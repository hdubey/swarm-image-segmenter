#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math_functions.h>
#include "pso_cluster.h"

/*
 * Get euclidean distance between 2 pixels
 */
__host__ __device__ 
float devGetDistance(int *first, int *second)
{
	float total = 0.0f;

	for (int i = 0; i < DATA_DIM; i++)
	{
		int res = (first[i] - second[i]);
		total += res * res;
	}

	return sqrt(total);
}

/*
 * Get error for given centroids
 */
__host__ __device__ 
float devFitness(short* assignMat, int* datas, int* centroids, int data_size, int cluster_size)
{
	float total = 0.0f;

	for (int i = 0; i < cluster_size; i++)
	{
		float subtotal = 0.0f;

		for (int j = 0; j < data_size; j++)
		{
			if (assignMat[j] == i)
				subtotal += devGetDistance(&datas[j * DATA_DIM], &centroids[i * DATA_DIM]);
		}

		total += subtotal / data_size;
	}

	return total / cluster_size;
}

/*
 * Assign pixels to centroids
 */
__host__ __device__ 
void devAssignDataToCentroid(short *assignMat, int *datas, int *centroids, int data_size, int cluster_size)
{
	for (int i = 0; i < data_size; i++)
	{
		int nearestCentroidIdx = 0;
		float nearestCentroidDist = INF;

		for (int j = 0; j < cluster_size; j++)
		{
			float nearestDist = devGetDistance(&datas[i * DATA_DIM], &centroids[j * DATA_DIM]);

			if (nearestDist < nearestCentroidDist)
			{
				nearestCentroidDist = nearestDist;
				nearestCentroidIdx = j;
			}
		}

		assignMat[i] = nearestCentroidIdx;
	}
}

/*
 * Initialize necessary variables for PSO
 */
void initialize(int *positions, int *velocities, int *pBests, int *gBest, const data* datas, int data_size,
	int particle_size, int cluster_size)
{
	for (int i = 0; i < particle_size * cluster_size * DATA_DIM; i+= DATA_DIM)
	{
		int rand = round(getRandom(0, data_size - 1));

		for(int j = 0; j < DATA_DIM; j++)
		{
			positions[i + j] = datas[rand].info[j];
			pBests[i + j] = datas[rand].info[j];
			velocities[i + j] = 0;
		}
	}

	for(int i = 0; i < cluster_size * DATA_DIM; i++)
		gBest[i] = pBests[i];
}

/*
 * Kernel to update particle
 */
__global__ void kernelUpdateParticle(int *positions, int *velocities, int *pBests, int *gBest, short *posAssign,
	int* datas, float rp, float rg, int data_size, int particle_size, int cluster_size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int strideParticle = i * cluster_size * DATA_DIM;
	int strideAssign = i * data_size;

	if(i >= particle_size)
		return;

	for (int j = 0; j < cluster_size * DATA_DIM; j += DATA_DIM)
	{
		for (int k = 0; k < DATA_DIM; k++)
		{
			// Update particle velocity and position
			velocities[strideParticle + j + k] = (int)lroundf(OMEGA * velocities[strideParticle + j + k]
					+ c1 * rp * (pBests[strideParticle + j + k] - positions[strideParticle + j + k])
					+ c2 * rg * (gBest[j + k] - positions[strideParticle + j + k]));

			positions[strideParticle + j + k] += velocities[strideParticle + j + k];
		}
	}

	devAssignDataToCentroid(&posAssign[strideAssign], datas, &positions[strideParticle], data_size, cluster_size);
}

/*
 * Kernel to update pBests
 */
 __global__ void kernelUpdatePBest(int *positions, int *pBests, short *posAssign, short *pBestAssign, 
 	int *datas, int data_size, int particle_size, int cluster_size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int strideParticle = i * cluster_size * DATA_DIM;
	int strideAssign = i * data_size;

	if(i >= particle_size)
		return;

	// Update pBest
	if (devFitness(&posAssign[strideAssign], datas, &positions[strideParticle], data_size, cluster_size)
			< devFitness(&pBestAssign[strideAssign], datas, &pBests[strideParticle], data_size, cluster_size))
	{
		// Update pBest position
		for (int k = 0; k < cluster_size * DATA_DIM; k++)
			pBests[strideParticle + k] = positions[strideParticle + k];

		// Update pBest assignment matrix
		for(int k = 0; k < data_size; k++)
			pBestAssign[strideAssign + k] = posAssign[strideAssign + k];
	}
}

/*
 * Wrapper to initialize and running PSO on device
 */
extern "C" GBest devicePsoClustering(data *datas, int *flatDatas, int data_size, int particle_size, 
	int cluster_size, int max_iter)
{
	// Initialize host memory
	int *positions = new int[particle_size * cluster_size * DATA_DIM];
	int *velocities = new int[particle_size * cluster_size * DATA_DIM];
	int *pBests = new int[particle_size * cluster_size * DATA_DIM];
	int *gBest = new int[cluster_size * DATA_DIM];
	short *posAssign = new short[particle_size * data_size];
	short *pBestAssign = new short[particle_size * data_size];
	short *gBestAssign = new short[data_size];

	// Initialize assignment matrix to cluster 0
	for(int i = 0; i < particle_size * data_size; i++)
	{
		posAssign[i] = 0;
		pBestAssign[i] = 0;

		if(i < data_size)
			gBestAssign[i] = 0;
	}

	initialize(positions, velocities, pBests, gBest, datas, data_size, particle_size, cluster_size);

	// Initialize device memory
	int *devPositions, *devVelocities, *devPBests, *devGBest;
	short *devPosAssign, *devPBestAssign;
	int *devDatas;

	size_t size = sizeof(int) * particle_size * cluster_size * DATA_DIM;
	size_t assign_size = sizeof(short) * particle_size * data_size;

	hipMalloc((void**)&devPositions, size);
	hipMalloc((void**)&devVelocities, size);
	hipMalloc((void**)&devPBests, size);
	hipMalloc((void**)&devGBest, sizeof(int) * cluster_size * DATA_DIM);
	hipMalloc((void**)&devPosAssign, assign_size);
	hipMalloc((void**)&devPBestAssign, assign_size);
	hipMalloc((void**)&devDatas, sizeof(int) * data_size * DATA_DIM);

	// Copy data from host to device
	hipMemcpy(devPositions, positions, size, hipMemcpyHostToDevice);
	hipMemcpy(devVelocities, velocities, size, hipMemcpyHostToDevice);
	hipMemcpy(devPBests, pBests, size, hipMemcpyHostToDevice);
	hipMemcpy(devGBest, gBest, sizeof(int) * cluster_size * DATA_DIM, hipMemcpyHostToDevice);
	hipMemcpy(devPosAssign, posAssign, assign_size, hipMemcpyHostToDevice);
	hipMemcpy(devPBestAssign, pBestAssign, assign_size, hipMemcpyHostToDevice);
	hipMemcpy(devDatas, flatDatas, sizeof(int) * data_size * DATA_DIM, hipMemcpyHostToDevice);

	// Threads and blocks number
	int threads = 32;
	int blocks = (particle_size / threads) + 1;

	// Iteration
	for (int iter = 0; iter < max_iter; iter++)
	{
		cout << "Iteration-" << iter + 1 << endl;
		
		float rp = getRandomClamped();
		float rg = getRandomClamped();

		kernelUpdateParticle<<<blocks, threads>>>(devPositions, devVelocities, devPBests, devGBest, 
				devPosAssign, devDatas, rp, rg, data_size, particle_size, cluster_size);

		kernelUpdatePBest<<<blocks, threads>>>(devPositions, devPBests, devPosAssign, devPBestAssign, 
 				devDatas, data_size, particle_size, cluster_size);

		// Compute gBest on host
		hipMemcpy(pBests, devPBests, size, hipMemcpyDeviceToHost);
		hipMemcpy(pBestAssign, devPBestAssign, assign_size, hipMemcpyDeviceToHost);

		for(int i = 0; i < particle_size; i++)
		{
			// Get slice of array
			int strideParticle = i * cluster_size * DATA_DIM;
			int strideAssign = i * data_size;

			// Compare pBest and gBest
			if (devFitness(&pBestAssign[strideAssign], flatDatas, &pBests[strideParticle], data_size, cluster_size)
					< devFitness(gBestAssign, flatDatas, gBest, data_size, cluster_size))
			{
				// Update gBest position
				for (int k = 0; k < cluster_size * DATA_DIM; k++)
					gBest[k] = pBests[strideParticle + k];

				// Update gBest assignment matrix
				for(int k = 0; k < data_size; k++)
					gBestAssign[k] = pBestAssign[strideAssign + k];
			}
		}

		hipMemcpy(devGBest, gBest, sizeof(int) * cluster_size * DATA_DIM, hipMemcpyHostToDevice);
	}

	// Copy gBest from device to host
	hipMemcpy(gBest, devGBest, sizeof(int) * cluster_size * DATA_DIM, hipMemcpyDeviceToHost);

	GBest gBestReturn;
	gBestReturn.gBestAssign = gBestAssign;

	return gBestReturn;
}